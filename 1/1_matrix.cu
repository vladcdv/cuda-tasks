#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void transpose_gpu(float *matFrom, float *matTo, int rows, int columns)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < columns && row < rows)
        matTo[col * rows + row] = matFrom[row * columns + col];
}

__global__ void transpose_gpu_coalesced(float *matFrom, float *matTo, int rows, int columns)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int col = idx % columns;
    int row = idx / columns;

    if (col < columns && row < rows)
        matTo[col * rows + row] = matFrom[row * columns + col];
}

void transpose_cpu(float *matFrom, float *matTo, int rows, int columns)
{
    // Iterate over input matrix and populate output matrix
    for (int row = 0; row < rows; ++row)
        for (int col = 0; col < columns; ++col)
        {
            matTo[col * rows + row] = matFrom[row * columns + col];
        }
}

inline hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    }
    return result;
}

void checkAsyncError()
{
    hipError_t syncErr = hipGetLastError();
    if (syncErr != hipSuccess)
    {
        printf("Kernel launch error: %s\n", hipGetErrorString(syncErr));
    }
}

void verify(float *a, float *b, int elCount)
{
    for (int i = 0; i < elCount; i++)
    {
        if (a[i] != b[i])
        {
            printf("Error! Matrices don't match!");
            return;
        }
    }
}

int main()
{
    clock_t start, end;

    const int ROWS = 2048 * 16;
    const int COLUMNS = 1024 * 16;
    const size_t TOTAL_SIZE = sizeof(float) * ROWS * COLUMNS;

    float *inputMatrixReference = (float *)malloc(TOTAL_SIZE);
    float *outputMatrixReference = (float *)malloc(TOTAL_SIZE);

    // Init host matrix with random data
    for (int row = 0; row < ROWS; ++row)
    {
        for (int col = 0; col < COLUMNS; ++col)
        {
            inputMatrixReference[row * COLUMNS + col] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
        }
    }

    {
        // Transpose on CPU
        start = clock();
        transpose_cpu(inputMatrixReference, outputMatrixReference, ROWS, COLUMNS);
        end = clock();
        printf("CPU transpose - %f seconds\n", double(end - start) / CLOCKS_PER_SEC);
    }

    {
        // Transpose on the GPU (no unified memory)
        start = clock();
        float *inputMatrixDevice;
        float *outputMatrixDevice;
        float *outputMatrixHost = (float *)malloc(TOTAL_SIZE);
        hipMalloc(&inputMatrixDevice, TOTAL_SIZE);
        hipMalloc(&outputMatrixDevice, TOTAL_SIZE);

        hipMemcpy(inputMatrixDevice, inputMatrixReference, TOTAL_SIZE, hipMemcpyHostToDevice);

        dim3 threads_per_block(16, 16, 1);
        dim3 number_of_blocks((COLUMNS + threads_per_block.x - 1) / threads_per_block.x, (ROWS + threads_per_block.y - 1) / threads_per_block.y, 1);
        transpose_gpu<<<number_of_blocks, threads_per_block>>>(inputMatrixDevice, outputMatrixDevice, ROWS, COLUMNS);
        hipMemcpy(outputMatrixHost, outputMatrixDevice, TOTAL_SIZE, hipMemcpyDeviceToHost);
        checkCuda(hipDeviceSynchronize());
        end = clock();

        // Verify correctness & performance
        checkAsyncError();
        verify(outputMatrixHost, outputMatrixReference, ROWS * COLUMNS);
        printf("GPU transpose (no UM) - %f seconds\n", double(end - start) / CLOCKS_PER_SEC);

        // Free resources
        hipFree(inputMatrixDevice);
        hipFree(outputMatrixDevice);
        free(outputMatrixHost);
        checkCuda(hipDeviceSynchronize());
    }

    {
        // Transpose on the GPU (unified memory)
        start = clock();
        float *inputMatrixDevice;
        float *outputMatrixDevice;
        hipMallocManaged(&inputMatrixDevice, TOTAL_SIZE);
        hipMallocManaged(&outputMatrixDevice, TOTAL_SIZE);

        hipMemcpy(inputMatrixDevice, inputMatrixReference, TOTAL_SIZE, hipMemcpyHostToDevice);

        dim3 threads_per_block(16, 16, 1);
        dim3 number_of_blocks((COLUMNS + threads_per_block.x - 1) / threads_per_block.x, (ROWS + threads_per_block.y - 1) / threads_per_block.y, 1);
        transpose_gpu<<<number_of_blocks, threads_per_block>>>(inputMatrixDevice, outputMatrixDevice, ROWS, COLUMNS);
        checkCuda(hipDeviceSynchronize());
        end = clock();

        // Verify correctness & performance
        checkAsyncError();
        verify(outputMatrixDevice, outputMatrixReference, ROWS * COLUMNS);
        printf("GPU transpose (UM) - %f seconds\n", double(end - start) / CLOCKS_PER_SEC);

        // Free resources
        hipFree(inputMatrixDevice);
        hipFree(outputMatrixDevice);
        checkCuda(hipDeviceSynchronize());
    }

    {
        // Transpose on the GPU (unified memory + coalesced memory access)
        start = clock();
        float *inputMatrixDevice;
        float *outputMatrixDevice;
        hipMallocManaged(&inputMatrixDevice, TOTAL_SIZE);
        hipMallocManaged(&outputMatrixDevice, TOTAL_SIZE);

        hipMemcpy(inputMatrixDevice, inputMatrixReference, TOTAL_SIZE, hipMemcpyHostToDevice);

        int threads_per_block = 128;
        int number_of_blocks = (ROWS * COLUMNS + threads_per_block - 1) / threads_per_block;
        transpose_gpu_coalesced<<<number_of_blocks, threads_per_block>>>(inputMatrixDevice, outputMatrixDevice, ROWS, COLUMNS);
        checkCuda(hipDeviceSynchronize());
        end = clock();

        // Verify correctness & performance
        checkAsyncError();
        verify(outputMatrixDevice, outputMatrixReference, ROWS * COLUMNS);
        printf("GPU transpose (UM + coalesced) - %f seconds\n", double(end - start) / CLOCKS_PER_SEC);

        // Free resources
        hipFree(inputMatrixDevice);
        hipFree(outputMatrixDevice);
        checkCuda(hipDeviceSynchronize());
    }

    {
        // Transpose on the GPU (no unified memory+ coalesced)
        start = clock();
        float *inputMatrixDevice;
        float *outputMatrixDevice;
        float *outputMatrixHost = (float *)malloc(TOTAL_SIZE);
        hipMalloc(&inputMatrixDevice, TOTAL_SIZE);
        hipMalloc(&outputMatrixDevice, TOTAL_SIZE);

        hipMemcpy(inputMatrixDevice, inputMatrixReference, TOTAL_SIZE, hipMemcpyHostToDevice);

        int threads_per_block = 128;
        int number_of_blocks = (ROWS * COLUMNS + threads_per_block - 1) / threads_per_block;
        transpose_gpu_coalesced<<<number_of_blocks, threads_per_block>>>(inputMatrixDevice, outputMatrixDevice, ROWS, COLUMNS);
        hipMemcpy(outputMatrixHost, outputMatrixDevice, TOTAL_SIZE, hipMemcpyDeviceToHost);
        checkCuda(hipDeviceSynchronize());
        end = clock();

        // Verify correctness & performance
        checkAsyncError();
        verify(outputMatrixHost, outputMatrixReference, ROWS * COLUMNS);
        printf("GPU transpose (no UM + coalesced) - %f seconds\n", double(end - start) / CLOCKS_PER_SEC);

        // Free resources
        hipFree(inputMatrixDevice);
        hipFree(outputMatrixDevice);
        free(outputMatrixHost);
        checkCuda(hipDeviceSynchronize());
    }

    free(inputMatrixReference);
    free(outputMatrixReference);
}
